
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>

namespace CudaHelpers {

	template<typename T>
	bool copy_vector_to_gpu(T* gpu_mem, const std::vector<T>& vec){
		hipError_t err;
		err = hipMalloc((void**)&gpu_mem, vec.size() * (size_t)sizeof(T));
		hipMemcpy((void*)gpu_mem, (void*)vec.data(), vec.size() * (size_t)sizeof(T), hipMemcpyHostToDevice);

		return err;
	}

	template<typename T>
	bool retrieve_vector_from_gpu(T* gpu_mem, const std::vector<T>& vec) {
		return hipMemcpy((void*)vec.data(), gpu_mem, vec.size() * (size_t)sizeof(T), hipMemcpyDeviceToHost);
	}
}

